
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand.h>

#define N (1 << 8)

__global__ void gemm_baseline(float *A, float *B, float *C)
{
    // Compute matrix multiplication
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (int k = 0; k < N; ++k)
    {
        sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
}

void gemm_verify(float *A, float *B, float *C)
{
    float *baseline = (float *)malloc(N * N * sizeof(float));
    for (int i = 0; i < N * N; i++)
        baseline[i] = 0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
            {
                baseline[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }

    for (int i = 0; i < N * N; i++)
    {
        if (abs(C[i] - baseline[i]) > 1e-3)
        {
            printf("fail: C[%d] = %f, baseline[%d] = %f\n", i, C[i], i, baseline[i]);
            break;
        }
    }
    free(baseline);
}

int main()
{
    float *A;
    float *B;
    float *C;

    // malloc A, B, C
    A = (float *)malloc(N * N * sizeof(float));
    B = (float *)malloc(N * N * sizeof(float));
    C = (float *)malloc(N * N * sizeof(float));

    // random initialize A, B
    for (int i = 0; i < N * N; ++i)
    {
        A[i] = (float)rand() / RAND_MAX;
        B[i] = (float)rand() / RAND_MAX;
    }

    // cumalloc A, B, C
    float *cu_A;
    float *cu_B;
    float *cu_C;
    hipMalloc((void **)&cu_A, N * N * sizeof(float));
    hipMalloc((void **)&cu_B, N * N * sizeof(float));
    hipMalloc((void **)&cu_C, N * N * sizeof(float));

    // copy from CPU to GPU
    hipMemcpy(cu_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cu_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // define gridsize and blocksize
    dim3 blocksize(4, 4);
    dim3 gridsize((N + blocksize.x - 1) / blocksize.x, (N + blocksize.y - 1) / blocksize.y);

    // compute
    clock_t start_time = clock();
    gemm_baseline<<<gridsize, blocksize>>>(cu_A, cu_B, cu_C);
    clock_t end_time = clock();
    double cpu_time_used = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    printf("运行时间：%f秒\n", cpu_time_used);

    // Copy from GPU to CPU
    hipMemcpy(C, cu_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // verify the result
    gemm_verify(A, B, C);

    // free mem
    hipFree(cu_A);
    hipFree(cu_B);
    hipFree(cu_C);

    free(A);
    free(B);
    free(C);

    return 0;
}
